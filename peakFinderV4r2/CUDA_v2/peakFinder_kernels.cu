#include "hip/hip_runtime.h"
#include "peakFinder.h"
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <functional>
float *d_data = NULL;
uint *d_conmap = NULL;
uint *d_centers = NULL;
uint *d_dense_centers = NULL;
Peak *d_peaks = NULL;

const int FILTER_PATCH_WIDTH = 32;
const int FILTER_PATCH_HEIGHT = 4; 
const int FILTER_THREADS_PER_PATCH = FILTER_PATCH_WIDTH * FILTER_PATCH_HEIGHT;
const int FILTER_PATCH_ON_WIDTH = (WIDTH) / FILTER_PATCH_WIDTH;
const int FILTER_PATCH_ON_HEIGHT = (HEIGHT + FILTER_PATCH_HEIGHT - 1) / FILTER_PATCH_HEIGHT;
const int FILTER_PATCH_PER_IMAGE = FILTER_PATCH_ON_WIDTH * FILTER_PATCH_ON_HEIGHT;
__global__ void filterByThrHigh_v2(const float *d_data, uint *d_centers)
{
	uint imgId = blockIdx.x / FILTER_PATCH_PER_IMAGE;
	uint patch_id = blockIdx.x % FILTER_PATCH_PER_IMAGE;
	uint patch_x = patch_id % FILTER_PATCH_ON_WIDTH;
	uint patch_y = patch_id / FILTER_PATCH_ON_WIDTH;
	__shared__ float data[FILTER_PATCH_HEIGHT * FILTER_PATCH_WIDTH];
	__shared__ uint idxs[FILTER_PATCH_HEIGHT * FILTER_PATCH_WIDTH];
	int irow = threadIdx.x / FILTER_PATCH_WIDTH;
	int icol = threadIdx.x % FILTER_PATCH_WIDTH;
	int row = patch_y * FILTER_PATCH_HEIGHT + irow;
	int col = patch_x * FILTER_PATCH_WIDTH + icol;
	const int NUM_NMS_AREA = FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT;
	int local_area = icol / FILTER_PATCH_HEIGHT;
	int local_pos = local_area * (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT) + irow * FILTER_PATCH_HEIGHT + icol % FILTER_PATCH_HEIGHT;
	uint device_pos = imgId * (WIDTH * HEIGHT) + row * WIDTH + col;
	__shared__ bool has_candidate[NUM_NMS_AREA];
	if (threadIdx.x < NUM_NMS_AREA) has_candidate[threadIdx.x] = false;
	__syncthreads();
	// load data
	if (row < WIDTH && col < HEIGHT){
		data[local_pos] = d_data[device_pos];
		idxs[local_pos] = device_pos;
	}
	else{
		data[local_pos] = 0;
	}

	if (data[local_pos] > thr_high)
		has_candidate[local_area] = true;
	__syncthreads();
	// find maximum
	local_area = threadIdx.x / (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	if (!has_candidate[local_area])
		return;
	const int local_tid = threadIdx.x % (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	const int local_offset =  local_area * (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	int num_of_working_threads = (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT) / 2;
	// if (local_tid >= num_of_working_threads) return;
	int idx_mul = 1;
	while (num_of_working_threads > 1 && local_tid < num_of_working_threads)
	{
		int idx1 = (local_tid * 2) * idx_mul + local_offset;
		int idx2 = idx1 + idx_mul;
		int idxm = data[idx1] > data[idx2] ? idx1 : idx2;
		data[idx1] = data[idxm];
		idxs[idx1] = idxs[idxm];
		__syncthreads();
		idx_mul *= 2;
		num_of_working_threads /= 2;
	}
	if (local_tid == 0)
	{
		uint write_pos = blockIdx.x * NUM_NMS_AREA + local_area;
		d_centers[write_pos] = idxs[local_offset];
	}
}

const int PATCH_WIDTH = (2 * HALF_WIDTH + 1);
const int FF_LOAD_THREADS_PER_CENTER = 64;
const int FF_THREADS_PER_CENTER = 32;
const int FF_INFO_THREADS_PER_CENTER = FF_THREADS_PER_CENTER;
// const int FF_SIDE_WIDTH = FF_THREADS_PER_CENTER / 4;
// const int FF_SIDE_OFFSET = 1 - FF_SIDE_WIDTH / 2;
const int FF_THREADS_PER_BLOCK = 64;
const int FF_LOAD_PASS = (2 * HALF_WIDTH + 1) * (2 * HALF_WIDTH + 1) / FF_LOAD_THREADS_PER_CENTER + 1;
const int FF_CENTERS_PER_BLOCK = FF_THREADS_PER_BLOCK / FF_LOAD_THREADS_PER_CENTER;

__device__ void calPreSum(int *preSum)
{
	for(int i = 1; i < FF_INFO_THREADS_PER_CENTER; i*=2){
		int prevIdx = threadIdx.x - i;
		int sum = preSum[threadIdx.x];
		if (prevIdx > 0){
			sum += preSum[prevIdx];
		}
		__syncthreads();
		preSum[threadIdx.x] = sum;
		__syncthreads();
	}
}

typedef float (*reducer) (const float &, const float &); 
typedef float (*transformer) (const int &);

__device__ float deviceAdd(const float &a, const float &b) {return a+b;}
__device__ float deviceMin(const float &a, const float &b) {return a<b?a:b;}
__device__ float deviceMax(const float &a, const float &b) {return a>b?a:b;}

const int WARP_SIZE = 32;
__device__ __inline__ float warpReduce(float val, int npix, reducer r)
{
	int offset = 32;
	if (npix < 32)
	{
		if (npix > 16) offset = 16;
		else if (npix > 8) offset = 8;
		else if (npix > 4) offset = 4;
		else if (npix > 2) offset = 2;
		else if (npix > 1) offset = 1;
		else offset = 0;
	}
	for(; offset > 0; offset /= 2){
		int srcIdx = threadIdx.x + offset;
		float nVal = __shfl_down(val, offset);
		if (srcIdx < npix){
			val = r(val, nVal);
		}
	}
	return val;
}
__device__ __inline__ float blockReduce(int npix, transformer t, reducer r)
{
	const int FF_PIX_PASS = (npix + FF_INFO_THREADS_PER_CENTER - 1) / FF_INFO_THREADS_PER_CENTER;
	__shared__ float buffer[32];
	for(int i = 0; i < FF_PIX_PASS; i++){
		uint tmp_id = i * FF_INFO_THREADS_PER_CENTER + threadIdx.x;
		int n = WARP_SIZE;
		if (i == FF_PIX_PASS - 1){
			n = npix % WARP_SIZE;
		}
		float val = warpReduce(t(tmp_id), n, r);
		if (threadIdx.x == 0){
			buffer[i] = val;
		}
	}
	return warpReduce(buffer[threadIdx.x], FF_PIX_PASS, r);
}

__device__ __inline__ bool inRing(int dr, int dc)
{
	float dist2 = dr * dr + dc * dc;
	const float lower = r0 * r0;
	const float upper = (r0 + dr) * (r0 + dr);
	return dist2 >= lower && dist2 <= upper;
}

__device__ __inline__ bool peakIsPreSelected(float son, float npix, float amp_max, float amp_tot)
{
	if (son < peak_son_min) return false;
	if (npix < peak_npix_min) return false;
	if (npix > peak_npix_max) return false;
	if (amp_max < peak_amax_thr) return false;
	if (amp_tot < peak_atot_thr) return false;
	return true; 
}
// one center per block
__global__ void floodFill_v2(const float *d_data, const uint *d_centers, Peak *d_peaks, uint *d_conmap)
{
	const uint center_id = d_centers[blockIdx.x];
	const uint img_id = center_id / (WIDTH * HEIGHT);
	const uint crow = center_id / WIDTH % HEIGHT;
	const uint ccol = center_id % WIDTH;
	__shared__ float data[PATCH_WIDTH][PATCH_WIDTH];
	__shared__ uint status[PATCH_WIDTH][PATCH_WIDTH];
	// load data
	for (int i = 0; i < FF_LOAD_PASS; i++)
	{
		const uint tmp_id = i * FF_LOAD_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		const int drow = crow + irow - HALF_WIDTH;
		const int dcol = ccol + icol - HALF_WIDTH;
		if (drow >= 0 && drow < HEIGHT && dcol >= 0 && dcol < WIDTH)
		{
			data[irow][icol] = d_data[img_id * (WIDTH * HEIGHT) + drow * WIDTH + dcol];
		}
		else if(irow < PATCH_WIDTH)
		{
			data[irow][icol] = 0;
		}
	}
	for(int i = 0; i < FF_LOAD_PASS; i++)
	{
		const uint tmp_id = i * FF_LOAD_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		if (irow < PATCH_WIDTH){
			status[irow][icol] = 0;
		}
		if (irow == HALF_WIDTH && icol == HALF_WIDTH){
			status[irow][icol] = center_id;
		}
	}
	__syncthreads();
	if (threadIdx.x >= FF_THREADS_PER_CENTER)
		return;
	// flood fill
	const int FF_SCAN_LENGTH = FF_THREADS_PER_CENTER / 8;
	const int sign_x[8] = {-1, 1, 1, -1, 1, 1, -1, -1};
	const int sign_y[8] = {1, 1, -1, -1, 1, -1, -1, 1};
	const int scanline_id = threadIdx.x / FF_SCAN_LENGTH;
	const int id_in_grp = threadIdx.x % (2 * FF_SCAN_LENGTH);
	const int base_v = id_in_grp - FF_SCAN_LENGTH;
	int icol = base_v * sign_x[scanline_id] + HALF_WIDTH;
	int irow = base_v * sign_y[scanline_id] + HALF_WIDTH;
	const int scangrp_id = threadIdx.x / (2 * FF_SCAN_LENGTH);
	const int dxs[4] = {-1, 1, 0, 0};
	const int dys[4] = {0, 0, 1, -1};
	const int dx = dxs[scangrp_id];
	const int dy = dys[scangrp_id];
	const float center_intensity = data[HALF_WIDTH][HALF_WIDTH];
	__shared__ bool is_local_maximum;
	is_local_maximum = true;
	for(int i = 1; i <= rank; i++){
		__syncthreads();
		if (!is_local_maximum) return;
		icol += dx;
		irow += dy;
		if (data[irow][icol] > center_intensity){
			is_local_maximum = false;
		}
		if (data[irow][icol] > thr_low){
			if (status[irow-dy][icol-dx] == center_id){
				status[irow][icol] = center_id;
			}
		}
	}

	const int bound = base_v > 0 ? base_v : -base_v;
	for(int i = 1; i <= FF_SCAN_LENGTH - 1; i++){
		__syncthreads();
		if (!is_local_maximum) return;
		if (i > bound) continue;
		icol += dx;
		irow += dy;
		// if (irow >= PATCH_WIDTH || icol >= PATCH_WIDTH || irow < 0 || icol < 0){
		// 	printf("irow:%d, icol:%d, img_id:%d, i:%d\n", irow, icol, img_id, i);
		// }
		if (data[irow][icol] > center_intensity){
			is_local_maximum = false;
		}
		if (data[irow][icol] > thr_low){
			if (status[irow-dy][icol-dx] == center_id){
				status[irow][icol] = center_id;
			}
		}
	} 

	const int FF_PROC_PASS = (PATCH_WIDTH * PATCH_WIDTH + FF_INFO_THREADS_PER_CENTER - 1) / FF_INFO_THREADS_PER_CENTER;
	// calculate peak info
	__shared__ float peak_data[PATCH_WIDTH * PATCH_WIDTH];
	__shared__ int peak_row[PATCH_WIDTH * PATCH_WIDTH];
	__shared__ int peak_col[PATCH_WIDTH * PATCH_WIDTH];
	// data compaction
	__shared__ int preSum[FF_INFO_THREADS_PER_CENTER];
	preSum[threadIdx.x] = 0;
	for(int i = 0; i < FF_PROC_PASS; i++){
		const uint tmp_id = i * FF_INFO_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		if (irow < PATCH_WIDTH && status[irow][icol] == center_id){
			preSum[threadIdx.x] += 1;
		}
	}
	calPreSum(preSum);
	int npix = preSum[FF_INFO_THREADS_PER_CENTER - 1];
	int counter = 0;
	__shared__ float bg_avg;
	__shared__ float bg_rms;
	__shared__ float bg_npix;
	if (threadIdx.x == 0){
		bg_avg = 0;
		bg_rms = 0;
		bg_npix = 0;
	}
	for(int i = 0; i < FF_PROC_PASS; i++){
		const uint tmp_id = i * FF_INFO_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		if (irow < PATCH_WIDTH){
			if (status[irow][icol] == center_id){
				int pos = counter;
				if (threadIdx.x > 0)
					pos += preSum[threadIdx.x - 1];
				peak_data[pos] = data[irow][icol];
				peak_row[pos] = irow;
				peak_col[pos] = icol;
				counter ++;
			}
			// calculate background info
			if (inRing(irow-HALF_WIDTH,icol-HALF_WIDTH) && data[irow][icol] < thr_low){
				float d = data[irow][icol];
				atomicAdd(&bg_avg,d);
				atomicAdd(&bg_rms,d * d);
				atomicAdd(&bg_npix, 1);
			}
		}
	}
	const int FF_PIX_PASS = (npix + FF_INFO_THREADS_PER_CENTER - 1) / FF_INFO_THREADS_PER_CENTER;
	__shared__ float buffer[32];
	#define BLOCK_REDUCE(v,t,r) \
	for(int i = 0; i < FF_PIX_PASS; i++){ \
		uint tmp_id = i * FF_INFO_THREADS_PER_CENTER + threadIdx.x; \
		int n = WARP_SIZE; \
		if (i == FF_PIX_PASS - 1){ \
			n = npix % WARP_SIZE; \
		} \
		float val = warpReduce(t(tmp_id), n, r); \
		if (threadIdx.x == 0){ \
			buffer[i] = val; \
		} \
	} \
	v= warpReduce(buffer[threadIdx.x], FF_PIX_PASS, r); 
	float samp; BLOCK_REDUCE(samp, [=]__device__(const int &id) -> float {return peak_data[id];}, deviceAdd);
	__shared__ Peak peak;
	if (threadIdx.x == 0)
	{
		bg_avg /= bg_npix;
		bg_rms = bg_rms / bg_npix - bg_avg * bg_avg;
		bg_rms = sqrtf(bg_rms);
		float noise_tot = bg_rms * sqrtf(npix);
		peak.amp_tot = samp - bg_avg * npix;
		peak.amp_max = center_intensity - bg_avg;
		peak.son = noise_tot > 0 ? peak.amp_tot / noise_tot : 0;
		peak.bkgd = bg_avg;
		peak.noise = bg_rms;
		peak.valid = peakIsPreSelected(peak.son, npix, peak.amp_max, peak.amp_tot);
	}
	__syncthreads();
	if (!peak.valid) return;
	// if (center_id == 4734401 && threadIdx.x == 0)
	// 	printf("sum of intensity:%f,\n", samp);
	float rmin; BLOCK_REDUCE(rmin, [=]__device__(const int &id) -> float {return peak_row[id];}, deviceMin);
	float rmax; BLOCK_REDUCE(rmax, [=]__device__(const int &id) -> float {return peak_row[id];}, deviceMax);
	float cmin; BLOCK_REDUCE(cmin, [=]__device__(const int &id) -> float {return peak_col[id];}, deviceMin);
	float cmax; BLOCK_REDUCE(cmax, [=]__device__(const int &id) -> float {return peak_col[id];}, deviceMax);
	float sar1; BLOCK_REDUCE(sar1, [=]__device__(const int &id) -> float {return peak_data[id] * peak_row[id];}, deviceAdd);
	float sac1; BLOCK_REDUCE(sac1, [=]__device__(const int &id) -> float {return peak_data[id] * peak_col[id];}, deviceAdd);
	float sar2; BLOCK_REDUCE(sar2, [=]__device__(const int &id) -> float {return peak_data[id] * peak_row[id] * peak_row[id];}, deviceAdd);
	float sac2; BLOCK_REDUCE(sac2, [=]__device__(const int &id) -> float {return peak_data[id] * peak_col[id] * peak_col[id];}, deviceAdd);
	if(threadIdx.x == 0){
		peak.evt = img_id / SHOTS;
		peak.seg = img_id % SHOTS;
		// printf("center_id: %d, evt: %f, seg:%f\n", center_id, peak.evt, peak.seg);
		peak.row = crow;
		peak.col = ccol;
		peak.npix = npix;
		peak.row_min = rmin;
		peak.row_max = rmax;
		peak.col_min = cmin;
		peak.col_max = cmax;
		if (samp > 0){
			sar1 = sar1 / samp;
			sac1 = sac1 / samp;
			sar2 = sar2 / samp - sar1 * sar1;
			sac2 = sac2 / samp - sac1 * sac1;
			peak.row_cgrav = sar1;
			peak.col_cgrav = sac1;
			peak.row_sigma = (npix > 1 && sar2 > 0) ? sqrtf(sar2) : 0;
			peak.col_sigma = (npix > 1 && sac2 > 0) ? sqrtf(sac2) : 0;
		}
		else {
			peak.row_cgrav = crow;
			peak.col_cgrav = ccol;
			peak.row_sigma = 0;
			peak.col_sigma = 0;
		}
		d_peaks[blockIdx.x] = peak;
	}

	// printf("center_id:%d\n", center_id);
	// write data
	for(int i = 0; i < FF_PROC_PASS; i++){
		const uint tmp_id = i * FF_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		const int drow = crow + irow - rank;
		const int dcol = ccol + icol - rank;
		if (irow < PATCH_WIDTH && status[irow][icol] == center_id && drow >= 0 && drow < HEIGHT && dcol >= 0 && dcol < WIDTH)
		{
			// if(img_id == 9)
			// 	printf("irow:%d, icol:%d, center_id:%d\n", irow, icol, center_id);
			d_conmap[img_id * (WIDTH * HEIGHT) + drow * WIDTH + dcol] = status[irow][icol];
		}
	}

}

struct is_center
{
	__device__
	bool operator()(const uint &x){
		// return x == addr_conmap[x];
		return x > 0;
	}
};

void checkCudaError(hipError_t err, const char* msg)
{
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code: %s\n", msg, hipGetErrorString(err));
	}
}

void getCudaError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code %s\n", msg, hipGetErrorString(err));
	}
}

void setUpData(float *h_data)
{
	checkCudaError(hipMalloc((void **)&d_data, LSIZE * sizeof(float)), "hipMalloc d_data");
	checkCudaError(hipMalloc((void **)&d_conmap, LSIZE * sizeof(uint)), "hipMalloc d_conmap");
	checkCudaError(hipMemset(d_conmap, 0, sizeof(uint)*LSIZE), "hipMemset d_conmap");
	checkCudaError(hipMemcpy(d_data, h_data, LSIZE * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy h2d");
}

void releaseData()
{
	hipFree(d_data);
	hipFree(d_conmap);
}


extern "C" void processImages(float *data, Peak *&peak_out, int &npeaks, uint *data_out)
{
	float miliseconds = 0.0f;
	hipEvent_t t0, t1;
	hipEventCreate(&t0);
	hipEventCreate(&t1);
	hipEventRecord(t0);
	setUpData(data);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("passing data to gpu takes %f miliseconds\n", miliseconds);

	// floodFill v2
	printf("filterByThrHigh_v2: num_blocks:%ld\n", FILTER_PATCH_PER_IMAGE * EVENTS * SHOTS);
	const int centers_size = FILTER_PATCH_PER_IMAGE * (FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT) * EVENTS * SHOTS;
	checkCudaError(hipMalloc((void **)&d_centers, centers_size * sizeof(uint)), "hipMalloc d_centers");
	checkCudaError(hipMemset(d_centers, 0, centers_size * sizeof(uint)), "hipMemset d_centers");
	checkCudaError(hipMalloc((void **)&d_dense_centers, centers_size * sizeof(uint)), "hipMalloc d_dense_centers");
	hipDeviceSynchronize();
	hipEventRecord(t0);
	filterByThrHigh_v2<<<FILTER_PATCH_PER_IMAGE * EVENTS * SHOTS, FILTER_THREADS_PER_PATCH>>>(d_data, d_centers);
	getCudaError("filterByThrHigh_v2");
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("filterByThrHigh_v2 takes %f miliseconds\n", miliseconds);
	hipEventRecord(t0);
	thrust::device_ptr<uint> dp_dense_centers = thrust::device_pointer_cast(d_dense_centers);
	thrust::device_ptr<uint> dp_centers = thrust::device_pointer_cast(d_centers);
	auto end_centers = thrust::copy_if(dp_centers, dp_centers + centers_size,  dp_dense_centers, is_center());
	int num_pix = end_centers - dp_dense_centers;
	printf("num of testing pixels:%d\n", num_pix);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("stream compaction takes %f miliseconds\n", miliseconds);

	const int NUM_BLOCKS = num_pix / FF_CENTERS_PER_BLOCK;
	npeaks = num_pix;

	hipEventRecord(t0);
	checkCudaError(hipMalloc((void **)&d_peaks, num_pix * sizeof(Peak)), "hipMalloc d_peaks");
	checkCudaError(hipMemset(d_peaks, 0, num_pix * sizeof(Peak)), "hipMemset d_peaks");
	floodFill_v2<<<NUM_BLOCKS, FF_THREADS_PER_BLOCK>>>(d_data, d_dense_centers, d_peaks, d_conmap);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("floodFill_v2 takes %f miliseconds\n", miliseconds);
	getCudaError("floodFill_v2");

	peak_out = new Peak[num_pix];
	hipEventRecord(t0);
	checkCudaError(hipMemcpy(peak_out, d_peaks, num_pix * sizeof(Peak), hipMemcpyDeviceToHost), "hipMemcpy d2h");
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("copying peaks to cpu takes %f miliseconds\n", miliseconds);
	if (data_out != NULL)
	{
		checkCudaError(hipMemcpy(data_out, d_conmap, LSIZE * sizeof(uint), hipMemcpyDeviceToHost), "hipMemcpy d2h");
	}

	releaseData();
}