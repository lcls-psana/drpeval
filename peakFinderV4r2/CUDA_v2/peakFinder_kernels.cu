#include "hip/hip_runtime.h"
#include "peakFinder.h"
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
float *d_data = NULL;
uint *d_conmap = NULL;
uint *d_centers = NULL;
uint *d_dense_centers = NULL;
Peak *d_peaks = NULL;

const int FILTER_PATCH_WIDTH = 32;
const int FILTER_PATCH_HEIGHT = 8; 
const int FILTER_THREADS_PER_PATCH = FILTER_PATCH_WIDTH * FILTER_PATCH_HEIGHT;
const int FILTER_PATCH_ON_WIDTH = (WIDTH) / FILTER_PATCH_WIDTH;
const int FILTER_PATCH_ON_HEIGHT = (HEIGHT + FILTER_PATCH_HEIGHT - 1) / FILTER_PATCH_HEIGHT;
const int FILTER_PATCH_PER_IMAGE = FILTER_PATCH_ON_WIDTH * FILTER_PATCH_ON_HEIGHT;
__global__ void filterByThrHigh_v2(const float *d_data, uint *d_centers)
{
	uint imgId = blockIdx.x / FILTER_PATCH_PER_IMAGE;
	uint patch_id = blockIdx.x % FILTER_PATCH_PER_IMAGE;
	uint patch_x = patch_id % FILTER_PATCH_ON_WIDTH;
	uint patch_y = patch_id / FILTER_PATCH_ON_WIDTH;
	__shared__ float data[FILTER_PATCH_HEIGHT * FILTER_PATCH_WIDTH];
	__shared__ uint idxs[FILTER_PATCH_HEIGHT * FILTER_PATCH_WIDTH];
	int irow = threadIdx.x / FILTER_PATCH_WIDTH;
	int icol = threadIdx.x % FILTER_PATCH_WIDTH;
	int row = patch_y * FILTER_PATCH_HEIGHT + irow;
	int col = patch_x * FILTER_PATCH_WIDTH + icol;
	const int NUM_NMS_AREA = FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT;
	int local_area = icol / FILTER_PATCH_HEIGHT;
	int local_pos = local_area * (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT) + irow * FILTER_PATCH_HEIGHT + icol % FILTER_PATCH_HEIGHT;
	uint device_pos = imgId * (WIDTH * HEIGHT) + row * WIDTH + col;
	__shared__ bool has_candidate[NUM_NMS_AREA];
	if (threadIdx.x < NUM_NMS_AREA) has_candidate[threadIdx.x] = false;
	__syncthreads();
	// load data
	if (row < WIDTH && col < HEIGHT){
		data[local_pos] = d_data[device_pos];
		idxs[local_pos] = device_pos;
	}
	else{
		data[local_pos] = 0;
	}

	if (data[local_pos] > thr_high)
		has_candidate[local_area] = true;
	__syncthreads();
	// find maximum
	local_area = threadIdx.x / (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	if (!has_candidate[local_area])
		return;
	const int local_tid = threadIdx.x % (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	const int local_offset =  local_area * (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT);
	int num_of_working_threads = (FILTER_PATCH_HEIGHT * FILTER_PATCH_HEIGHT) / 2;
	if (local_tid >= num_of_working_threads) return;
	int idx_mul = 1;
	while (num_of_working_threads > 1 && local_tid < num_of_working_threads)
	{
		int idx1 = (local_tid * 2) * idx_mul + local_offset;
		int idx2 = idx1 + idx_mul;
		int idxm = data[idx1] > data[idx2] ? idx1 : idx2;
		data[idx1] = data[idxm];
		idxs[idx1] = idxs[idxm];
		__syncthreads();
		idx_mul *= 2;
		num_of_working_threads /= 2;
	}
	if (local_tid < NUM_NMS_AREA)
	{
		uint write_pos = blockIdx.x * NUM_NMS_AREA + local_area;
		d_centers[write_pos] = idxs[local_offset];
	}
}

const int PATCH_WIDTH = (2 * rank + 1);
const int FF_LOAD_THREADS_PER_CENTER = 64;
const int FF_THREADS_PER_CENTER = 32;
// const int FF_SIDE_WIDTH = FF_THREADS_PER_CENTER / 4;
// const int FF_SIDE_OFFSET = 1 - FF_SIDE_WIDTH / 2;
const int FF_THREADS_PER_BLOCK = 64;
const int FF_LOAD_PASS = (2 * rank + 1) * (2 * rank + 1) / FF_LOAD_THREADS_PER_CENTER + 1;
const int FF_CENTERS_PER_BLOCK = FF_THREADS_PER_BLOCK / FF_LOAD_THREADS_PER_CENTER;
// one center per block
__global__ void floodFill_v2(const float *d_data, const uint *d_centers, uint *d_conmap)
{
	const uint center_id = d_centers[blockIdx.x];
	const uint img_id = center_id / (WIDTH * HEIGHT);
	const uint crow = center_id / WIDTH % HEIGHT;
	const uint ccol = center_id % WIDTH;
	__shared__ float data[PATCH_WIDTH][PATCH_WIDTH];
	__shared__ uint status[PATCH_WIDTH][PATCH_WIDTH];
	// load data
	for (int i = 0; i < FF_LOAD_PASS; i++)
	{
		const uint tmp_id = i * FF_LOAD_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		const int drow = crow + irow - rank;
		const int dcol = ccol + icol - rank;
		if (drow >= 0 && drow < HEIGHT && dcol >= 0 && dcol < WIDTH)
		{
			data[irow][icol] = d_data[img_id * (WIDTH * HEIGHT) + drow * WIDTH + dcol];
		}
		else if(irow < PATCH_WIDTH)
		{
			data[irow][icol] = 0;
		}
	}
	for(int i = 0; i < FF_LOAD_PASS; i++)
	{
		const uint tmp_id = i * FF_LOAD_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		if (irow < PATCH_WIDTH){
			status[irow][icol] = 0;
		}
		if (irow == rank && icol == rank){
			status[irow][icol] = 1;
		}
	}
	__syncthreads();
	if (threadIdx.x >= FF_THREADS_PER_CENTER)
		return;
	// flood fill
	const int FF_SCAN_LENGTH = FF_THREADS_PER_CENTER / 8;
	const int sign_x[8] = {-1, 1, 1, -1, 1, 1, -1, -1};
	const int sign_y[8] = {1, 1, -1, -1, 1, -1, -1, 1};
	const int scanline_id = threadIdx.x / FF_SCAN_LENGTH;
	const int base_v = threadIdx.x % (2 * FF_SCAN_LENGTH) - FF_SCAN_LENGTH;
	int icol = base_v * sign_x[scanline_id] + rank;
	int irow = base_v * sign_y[scanline_id] + rank;
	const int scangrp_id = threadIdx.x / (2 * FF_SCAN_LENGTH);
	const int dxs[4] = {-1, 1, 0, 0};
	const int dys[4] = {0, 0, 1, -1};
	const int dx = dxs[scangrp_id];
	const int dy = dys[scangrp_id];
	// __shared__ bool is_local_maximum;
	// is_local_maximum = true;
	for(int i = 1; i <= rank; i++){
		__syncthreads();
		icol += dx;
		irow += dy;
		if (data[irow][icol] > thr_low){
			if (status[irow-dy][icol-dx] > 0){
				status[irow][icol] = center_id;
			}
		}
	}
	// write data
	const int FF_WRITE_PASS = (PATCH_WIDTH * PATCH_WIDTH + FF_THREADS_PER_CENTER - 1) / FF_THREADS_PER_CENTER;
	for(int i = 0; i < FF_WRITE_PASS; i++){
		const uint tmp_id = i * FF_THREADS_PER_CENTER + threadIdx.x;
		const uint irow = tmp_id / PATCH_WIDTH;
		const uint icol = tmp_id % PATCH_WIDTH;
		const int drow = crow + irow - rank;
		const int dcol = ccol + icol - rank;
		if (irow < PATCH_WIDTH && status[irow][icol] > 0 && drow >= 0 && drow < HEIGHT && dcol >= 0 && dcol < WIDTH)
		{
			// if(img_id == 9)
			// 	printf("irow:%d, icol:%d, center_id:%d\n", irow, icol, center_id);
			d_conmap[img_id * (WIDTH * HEIGHT) + drow * WIDTH + dcol] = status[irow][icol];
		}
	}

}

struct is_center
{
	__device__
	bool operator()(const uint &x){
		// return x == addr_conmap[x];
		return x > 0;
	}
};

void checkCudaError(hipError_t err, const char* msg)
{
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code: %s\n", msg, hipGetErrorString(err));
	}
}

void getCudaError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code %s\n", msg, hipGetErrorString(err));
	}
}

void setUpData(float *h_data)
{
	checkCudaError(hipMalloc((void **)&d_data, LSIZE * sizeof(float)), "hipMalloc d_data");
	checkCudaError(hipMalloc((void **)&d_conmap, LSIZE * sizeof(uint)), "hipMalloc d_conmap");
	checkCudaError(hipMemset(d_conmap, 0, sizeof(uint)*LSIZE), "hipMemset d_conmap");
	checkCudaError(hipMemcpy(d_data, h_data, LSIZE * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy h2d");
}

void releaseData()
{
	hipFree(d_data);
	hipFree(d_conmap);
}


extern "C" void processImages(float *data, Peak *peak_out, uint *data_out)
{
	float miliseconds = 0.0f;
	hipEvent_t t0, t1;
	hipEventCreate(&t0);
	hipEventCreate(&t1);
	hipEventRecord(t0);
	setUpData(data);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("passing data to gpu takes %f miliseconds\n", miliseconds);

	// floodFill v2
	printf("filterByThrHigh_v2: num_blocks:%ld\n", FILTER_PATCH_PER_IMAGE * EVENTS * SHOTS);
	const int centers_size = FILTER_PATCH_PER_IMAGE * (FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT) * EVENTS * SHOTS;
	checkCudaError(hipMalloc((void **)&d_centers, centers_size * sizeof(uint)), "hipMalloc d_centers");
	checkCudaError(hipMemset(d_centers, 0, centers_size * sizeof(uint)), "hipMemset d_centers");
	checkCudaError(hipMalloc((void **)&d_dense_centers, centers_size * sizeof(uint)), "hipMalloc d_dense_centers");
	hipDeviceSynchronize();
	hipEventRecord(t0);
	filterByThrHigh_v2<<<FILTER_PATCH_PER_IMAGE * EVENTS * SHOTS, FILTER_THREADS_PER_PATCH>>>(d_data, d_centers);
	getCudaError("filterByThrHigh_v2");
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("filterByThrHigh_v2 takes %f miliseconds\n", miliseconds);
	hipEventRecord(t0);
	thrust::device_ptr<uint> dp_dense_centers = thrust::device_pointer_cast(d_dense_centers);
	thrust::device_ptr<uint> dp_centers = thrust::device_pointer_cast(d_centers);
	auto end_centers = thrust::copy_if(dp_centers, dp_centers + centers_size,  dp_dense_centers, is_center());
	int num_pix = end_centers - dp_dense_centers;
	printf("num of testing pixels:%d\n", num_pix);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("stream compaction takes %f miliseconds\n", miliseconds);

	const int NUM_BLOCKS = num_pix / FF_CENTERS_PER_BLOCK;

	hipEventRecord(t0);
	floodFill_v2<<<NUM_BLOCKS, FF_THREADS_PER_BLOCK>>>(d_data, d_dense_centers, d_conmap);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("floodFill_v2 takes %f miliseconds\n", miliseconds);
	getCudaError("floodFill_v2");

	if (data_out != NULL)
	{
		checkCudaError(hipMemcpy(data_out, d_conmap, LSIZE * sizeof(uint), hipMemcpyDeviceToHost), "hipMemcpy d2h");
	}

	releaseData();
}