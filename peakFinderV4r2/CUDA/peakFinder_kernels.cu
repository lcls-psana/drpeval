#include "hip/hip_runtime.h"
#include "peakFinder.h"
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
float *d_data = NULL;
uint *d_conmap = NULL;
uint *d_centers = NULL;
Peak *d_peaks = NULL;
bool *d_stop = NULL;
bool *d_acc_stop = NULL;
__device__ uint* addr_conmap = NULL;
// const uint WIN_TOP = 0;
// const uint WIN_BOT = HEIGHT;
// const uint WIN_LEFT = 0;
// const uint WIN_RIGHT = WIDTH;

__global__ void filterByThrHigh(const float *d_data, uint* d_conmap, uint imgId)
{
	uint pixId = (blockIdx.x % BLOCKS_PER_SHOT) * blockDim.x + threadIdx.x;
	// imgId = blockIdx.x / BLOCKS_PER_SHOT;
	if (pixId < WIDTH * HEIGHT)
	{
		uint dataId = imgId * WIDTH * HEIGHT + pixId;
		if (d_data[dataId] > thr_high)
			d_conmap[dataId] = dataId;
	}
}

__global__ void floodFill(const float *d_data, uint* d_conmap, bool *d_stop, bool *d_acc_stop, uint imgId)
{
	// imgId = blockIdx.x / BLOCKS_PER_SHOT;
	if (d_stop[imgId]) return;
	uint pixId = (blockIdx.x % BLOCKS_PER_SHOT) * blockDim.x + threadIdx.x;
	if (pixId < WIDTH * HEIGHT)
	{
		uint dataId = imgId * WIDTH * HEIGHT + pixId;
		float intensity = d_data[dataId];
		if (intensity > thr_low)
		{
			uint row = pixId / WIDTH, col = pixId % WIDTH;
			uint status = d_conmap[dataId];
			bool changed = false;
			if (col > 0)
			{
				uint tgt_Id = d_conmap[dataId-1];
				if (tgt_Id > 0 && (status == 0 || d_data[tgt_Id] > intensity) && col - tgt_Id % WIDTH <= rank)
				{
					d_conmap[dataId] = tgt_Id; changed = true;
				}
			}
			if (col < WIDTH - 1)
			{
				uint tgt_Id = d_conmap[dataId+1];
				if (tgt_Id > 0 && (status == 0 || d_data[imgId * WIDTH * HEIGHT + tgt_Id] > intensity) && tgt_Id % WIDTH - col <= rank)
				{
					d_conmap[dataId] = tgt_Id; changed = true;
				}
			}	
			if (row > 0)
			{
				uint tgt_Id = d_conmap[dataId-WIDTH];
				if (tgt_Id > 0 && (status == 0 || d_data[imgId * WIDTH * HEIGHT + tgt_Id] > intensity) && row - (tgt_Id / WIDTH) % HEIGHT <= rank)
				{
					d_conmap[dataId] = tgt_Id; changed = true;
				}
			}
			if (row < HEIGHT - 1)
			{
				uint tgt_Id = d_conmap[dataId+WIDTH];
				if (tgt_Id > 0 && (status == 0 || d_data[imgId * WIDTH * HEIGHT + tgt_Id] > intensity) && (tgt_Id / WIDTH) % HEIGHT - row <= rank)
				{
					d_conmap[dataId] = tgt_Id; changed = true;
				}
			}
			if (changed) d_acc_stop[imgId] = false;
		}
	}
}

void checkCudaError(hipError_t err, const char* msg)
{
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code: %s\n", msg, hipGetErrorString(err));
	}
}

void getCudaError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("failed: %s\n, error code %s\n", msg, hipGetErrorString(err));
	}
}

void setUpData(float *h_data)
{
	checkCudaError(hipMalloc((void **)&d_data, LSIZE * sizeof(float)), "hipMalloc d_data");
	checkCudaError(hipMalloc((void **)&d_conmap, LSIZE * sizeof(uint)), "hipMalloc d_conmap");
	checkCudaError(hipMalloc((void **)&d_stop, EVENTS * SHOTS * sizeof(bool)), "hipMalloc d_stop");
	checkCudaError(hipMalloc((void **)&d_acc_stop, EVENTS * SHOTS * sizeof(bool)), "hipMalloc d_acc_stop");
	checkCudaError(hipMemset(d_conmap, 0, sizeof(uint)*LSIZE), "hipMemset d_conmap");
	checkCudaError(hipMemset(d_acc_stop, false, sizeof(bool) * EVENTS * SHOTS), "hipMemset d_acc_stop");
	checkCudaError(hipMemcpy(d_data, h_data, LSIZE * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy h2d");
}

void releaseData()
{
	hipFree(d_data);
	hipFree(d_conmap);
}


extern "C" void processImages(float *data, Peak *peak_out, uint *data_out)
{
	float miliseconds = 0.0f;
	hipEvent_t t0, t1;
	hipEventCreate(&t0);
	hipEventCreate(&t1);
	hipEventRecord(t0);
	setUpData(data);
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("passing data to gpu takes %f miliseconds\n", miliseconds);
	hipEventRecord(t0);
	for(uint imgId = 0; imgId < EVENTS * SHOTS; imgId++)
	{
		filterByThrHigh<<<BLOCKS_PER_SHOT, THREADS_PER_BLOCK>>>(d_data, d_conmap, imgId);
		getCudaError("filterByThrHigh");
	}
	for (int i = 0; i < 2 * rank; i++)
	{
		checkCudaError(hipMemcpy(d_stop, d_acc_stop, sizeof(bool) * EVENTS * SHOTS, hipMemcpyDeviceToDevice), "hipMemcpy, d2d");
		checkCudaError(hipMemset(d_acc_stop, true, sizeof(bool) * EVENTS * SHOTS), "hipMemset d_acc_stop");	
		for (uint imgId = 0; imgId < EVENTS * SHOTS; imgId++)
		{
			floodFill<<<BLOCKS_PER_SHOT,THREADS_PER_BLOCK>>>(d_data, d_conmap, d_stop, d_acc_stop, imgId);
			getCudaError("floodFill");
		}
	}
	hipEventRecord(t1);
	hipEventSynchronize(t1);
	hipEventElapsedTime(&miliseconds, t0, t1);
	printf("processing takes %f miliseconds\n", miliseconds);
	if (data_out != NULL)
	{
		checkCudaError(hipMemcpy(data_out, d_conmap, LSIZE * sizeof(uint), hipMemcpyDeviceToHost), "hipMemcpy d2h");
	}

	releaseData();
}