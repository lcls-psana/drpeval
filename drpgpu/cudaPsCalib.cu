#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include <sys/time.h>
#include <iostream>
#include <iomanip>
using namespace std;

#include <string>
#include <sstream>
#include <fstream>

#include "cudaPsCalib.h"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__device__ void reduce(float *sdata)
{
  int tid = threadIdx.x;
  
  // do reduction in shared mem
  for(int s=1; s < blockDim.x; s*=2){
    
    int index = 2 * s * tid;

    if (index < blockDim.x) {
      // if a block is not a multiple of two, leave as-is
      if (index + s < blockDim.x)
        sdata[index] += sdata[index + s];
    }

    __syncthreads();
  }

}

/* -------------------------- calibration kernels ------------------------------*/

__global__ void kernel(float *a, 
                        int offset,
                        float *dark,
                        short *bad,
                        float cmmThr,
                        int streamSize,
                        float *blockSum, 
                        int *cnBlockSum){
  __shared__ float sdata[N_COLS];
  __shared__ float scount[N_COLS];

  int tid = threadIdx.x;
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  int iData = offset + idx;
  int iDark = iData % N_PIXELS;
  
  // subtrack pedestal and only use data with flag good pixel (bad==1)
  // and with value below a user-specified threshold.
  a[iData] -= dark[iDark];
  a[iData] *= bad[iDark];
  sdata[tid] = a[iData] * (a[iData] < cmmThr);
  scount[tid] = 1.0f * bad[iDark] * (a[iData] < cmmThr);

  __syncthreads();
 
  // calculate blocksum and blockcount
  reduce(sdata);
  reduce(scount);
  
  // save results - calculate block id using offset
  if (tid == 0){
    int iBlock = floor( (double) iData / blockDim.x );
    blockSum[iBlock] = sdata[0];
    cnBlockSum[iBlock] = (int)scount[0];
  }
  
}


__global__ void common_mode(float *blockSum, int *cnBlockSum, float *sectorSum, int *cnSectorSum, int offset)
{
  // calculate sector sum and sector count
  __shared__ float s_blockSum[N_ROWS];
  __shared__ float s_cnBlockSum[N_ROWS];

  int tid = threadIdx.x;
  int iBlock = tid + offset;
  s_blockSum[tid] = blockSum[iBlock];
  s_cnBlockSum[tid] = (float)cnBlockSum[iBlock];

  __syncthreads();

  reduce(s_blockSum);
  reduce(s_cnBlockSum);
  
  // save results - calculate sector id using offset
  if (tid == 0){
    int iSector = floor( (double) iBlock / blockDim.x );
    sectorSum[iSector] = s_blockSum[0];
    cnSectorSum[iSector] = (int)s_cnBlockSum[0];
  }
  
}

__global__ void common_mode_apply(float *a, float *sectorSum, int *cnSectorSum, float *gain, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  int iGain = i % N_PIXELS;
  int iSector = floor( (double) i / SECTOR_SIZE );
  a[i] = ( a[i] - (sectorSum[iSector] / cnSectorSum[iSector]) ) * gain[iGain];
}
   

/* ---------------------- host code -----------------------------*/
void write_file(string fileName, float *data, int n)
{
  FILE *pFile = fopen(fileName.c_str(), "w");
  if (pFile)
  {
    for (int i=0; i<n; i++)
      fprintf(pFile, "%f\n", data[i]);
  }
  fclose(pFile);
}

void fill( float *p, int n, float val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(float *aCalc, float *aKnown, int nEvents)
{
  float maxE = 0;
  for (int i = 0; i < nEvents; i++) {
    int offset = i * N_PIXELS;
    for (int j = 0; j < N_PIXELS; j++) {
      int idx = offset + j;
      float error = fabs(aCalc[idx]-aKnown[j]);
      //if (error > 5.0)
      //  printf("offset: %d j: %d idx: %d error %e aCalc[idx]: %8.2f aKnown[j]: %8.2f\n", offset, j, idx, error, aCalc[idx], aKnown[j]);
      if (error > maxE) maxE = error;
    }
  }
  return maxE;
}

// used in host_calculation qsort function
int compare (const void * a, const void * b)
{
  float fa = *(const float*) a;
  float fb = *(const float*) b;
  return (fa > fb) - (fa < fb);
}

// host-side calculation comparision
void host_calc(float *a, float *dark, float cmmThr) {
  // host calculation
  struct timeval start, end;

  long seconds, useconds;
  double mtime;

  gettimeofday(&start, NULL);
  
  // dark
  for(int i = 0; i < N_PIXELS; i++)
    a[i] -= dark[i];

  // common mode 
  float *sectorMedian = (float *)malloc(MAX_QUADS * MAX_SECTORS * sizeof(float));
  for (int i = 0; i < MAX_QUADS * MAX_SECTORS; i++) {
    
    int offset = i * SECTOR_SIZE;
    
    // select only this sector and sort this sector
    float *sector = (float *)malloc(SECTOR_SIZE * sizeof(float));
    for (int j = 0; j < SECTOR_SIZE; j++) {
      sector[j] = a[offset + j]; 
    }

    //printf("\n");
    //printf("s[0]=%6.2f, s[1]=%6.2f, s[2]=%6.2f\n", sector[0], sector[1], sector[2]);
    
    qsort(sector, SECTOR_SIZE, sizeof(float), compare);
    //printf("%6.2f, %6.2f, %6.2f ... %6.2f, %6.2f, %6.2f\n", sector[0], sector[1], sector[2], sector[SECTOR_SIZE-3], sector[SECTOR_SIZE-2], sector[SECTOR_SIZE-1]);
    
    // apply the threshold
    int foundPos = 0;
    for (int j = SECTOR_SIZE - 1; j >= 0; j--) {
      if (sector[j] <= cmmThr) {
        foundPos = j;
        break;
      }
      if (j == 0) foundPos = SECTOR_SIZE - 1;
    }   
    
    // calculate median
    if(foundPos%2 == 0) {
      sectorMedian[i] = (sector[foundPos/2] + sector[foundPos/2 - 1]) / 2.0;
    } else {
      sectorMedian[i] = sector[foundPos/2];
    } 
    free(sector);
    printf("sector: %d foundPos: %d med: %6.4f \n", i, foundPos, sectorMedian[i]); 
    
  }

  // apply common mode
  for(int i=0; i < N_PIXELS; i++) {
    int iSector = floor(i / SECTOR_SIZE);
    a[i] -= sectorMedian[iSector];
  }
  
  gettimeofday(&end, NULL);

  seconds  = end.tv_sec  - start.tv_sec;
  useconds = end.tv_usec - start.tv_usec;
  mtime = ((seconds) * 1000000 + useconds)/1000.0;// + 0.5;

  cout << "Host dark-subtraction and common mode took "<< mtime <<" ms for 1 event."<< endl;
}

int main(int argc, char **argv)
{
  const int nEvents = atoi(argv[1]);			        // no. of events
  const int n = N_PIXELS * nEvents;			          // total number of pixels
  
  const int blockSize = N_COLS;                   // block size is set to no. of columns in a sector

  const int bytes = n * sizeof(float);			      // total size (bytes)
  const int darkBytes = N_PIXELS * sizeof(float);	// dark size (bytes)

  const int nBlocks = N_ROWS * nEvents; 
  const int blockSumBytes = nBlocks * sizeof(float);

  const int nSectors = MAX_QUADS * MAX_SECTORS * nEvents;
  const int sectorSumBytes = nSectors * sizeof(float);

  const float cmmThr = 10.0f;
  
  int devId = 0;
  if (argc > 2) devId = atoi(argv[2]);			     // device ID (optional)
  
  // print device name
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  // RAW * nEVents
  float *a, *d_a; 						
  checkCuda( hipHostMalloc((void**)&a, bytes) );    // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) );  // device data only allocate enough for 1 evt
  // SINGLE RAW
  float *raw;                                               
  checkCuda( hipHostMalloc((void**)&raw, darkBytes) );     
  // PEDESTAL
  float *dark, *d_dark;					 
  checkCuda( hipHostMalloc((void**)&dark, darkBytes) ); 	
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) );		
  // PER-PIXEL GAIN
  float *gain, *d_gain;					 	
  checkCuda( hipHostMalloc((void**)&gain, darkBytes) ); 	
  checkCuda( hipMalloc((void**)&d_gain, darkBytes) );		
  // BAD PIXEL FLAGS
  short *bad, *d_bad;           
  checkCuda( hipHostMalloc((void**)&bad, N_PIXELS * sizeof(short)) );  
  checkCuda( hipMalloc((void**)&d_bad, N_PIXELS * sizeof(short)) );    
  // CALIBRATED
  float *calib, *d_calib;					 	
  checkCuda( hipHostMalloc((void**)&calib, darkBytes) ); 	
  checkCuda( hipMalloc((void**)&d_calib, darkBytes) );		  
  // Sum of each block
  float *d_blockSum, *blockSum; 
  checkCuda( hipMalloc((void**)&d_blockSum, blockSumBytes) );
  checkCuda( hipHostMalloc((void**)&blockSum, blockSumBytes) );
  hipMemset(d_blockSum, 0, blockSumBytes);
  int *d_cnBlockSum, *cnBlockSum;
  checkCuda( hipMalloc((void**)&d_cnBlockSum, nBlocks * sizeof(int)) );
  checkCuda( hipHostMalloc((void**)&cnBlockSum, nBlocks * sizeof(int)) );
  hipMemset(d_cnBlockSum, 0, nBlocks * sizeof(int));
  // Sum of each sector
  float *d_sectorSum, *sectorSum; 
  checkCuda( hipMalloc((void**)&d_sectorSum, sectorSumBytes) );
  checkCuda( hipHostMalloc((void**)&sectorSum, sectorSumBytes) );
  hipMemset(d_sectorSum, 0, sectorSumBytes);
  int * d_cnSectorSum;
  checkCuda( hipMalloc((void**)&d_cnSectorSum, nSectors * sizeof(int)) );
  // Peak centroids - allocate for all events
  // 8 centers per patch, 
  // 12x47=564  patches per sector, 
  // 564x8=4512 centers per sector
  // 4512x32 = 144384 centers per event.
  const int nCentersPerSector = FILTER_PATCH_PER_SECTOR * (FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT);
  const int nCentersPerEvent = nCentersPerSector * MAX_QUADS * MAX_SECTORS;
  const int nCenters = nCentersPerEvent * nEvents;
  uint *d_centers, *centers;
  checkCuda( hipMalloc((void**)&d_centers, nCenters * sizeof(uint)) );
  checkCuda( hipHostMalloc((void**)&centers, nCenters * sizeof(uint)) );
  checkCuda( hipMemset(centers, 0, nCenters * sizeof(uint)) );
  checkCuda( hipMemset(d_centers, 0, nCenters * sizeof(uint)));
  // Peaks - peak is allocated for all events since we need to copy
  // peaks for each event out.
  int nPeaks = nCenters;  
  Peak *d_peaks, *peaks;
  checkCuda( (hipMalloc((void**)&d_peaks, nPeaks * sizeof(Peak))) );
  checkCuda( (hipHostMalloc((void**)&peaks, nPeaks * sizeof(Peak))) );
  checkCuda( (hipMemset(d_peaks, 0, nPeaks * sizeof(Peak))) );
  checkCuda( (hipMemset(peaks, 0, nPeaks * sizeof(Peak))) );
  uint *d_conmap;
  checkCuda( (hipMalloc((void**)&d_conmap, n * sizeof(uint))) );
  checkCuda( (hipMemset(d_conmap, 0, n * sizeof(uint))) );

  //load the text file and put it into a single string:
  ifstream inR("data/cxid9114_r95_evt01_raw.txt");
  ifstream inP("data/cxid9114_r95_evt01_ped.txt");
  ifstream inG("data/cxid9114_r95_evt01_gmap.txt");
  ifstream inB("data/cxid9114_r95_evt01_stmask.txt"); // 0 - bad, 1 - Good
  ifstream inC("data/cxid9114_r95_evt01_calib.txt");
  
  // Fill arrays from text files
  string line;
  for (unsigned int i=0; i<N_PIXELS; i++){
    getline(inR, line);
    raw[i] = atof(line.c_str());
    getline(inP, line);
    dark[i] = atof(line.c_str());
    getline(inG, line);
    gain[i] = atof(line.c_str());
    getline(inB, line);
    bad[i] = atoi(line.c_str());
    getline(inC, line);
    calib[i] = atof(line.c_str());
    //populate all events with the same set of test data
    for (int j=0; j<nEvents; j++) {
      int offset = j * N_PIXELS;
      a[offset + i] = raw[i];
    }
  }
  puts("Input\n");
  printf("Data       : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Dark       : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", dark[0], dark[1], dark[2], dark[N_PIXELS-3], dark[N_PIXELS-2], dark[N_PIXELS-1]);
  printf("Bad pixels : %d %d %d...%d %d %d\n", bad[0], bad[1], bad[2], bad[N_PIXELS-3], bad[N_PIXELS-2], bad[N_PIXELS-1]);
  printf("Pixel gain : %8.2f %8.2f %8.2f ... %8.2f %8.2f %8.2f\n", gain[0], gain[1], gain[2], gain[N_PIXELS-3], gain[N_PIXELS-2], gain[N_PIXELS-1]);
  

  // host calculation 
  /*host_calc(raw, dark, cmmThr);

  
  printf("Host Calculation\n");
  printf("Input values (Data calc.): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", raw[0], raw[1], raw[2], raw[N_PIXELS-3], raw[N_PIXELS-2], raw[N_PIXELS-1]);
  printf("Input values (Data known): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", calib[0], calib[1], calib[2], calib[N_PIXELS-3], calib[N_PIXELS-2], calib[N_PIXELS-1]);
  printf("  max error: %e\n", maxError(raw, calib, 1));
  */

  // 
  // serial copy for one dark, bad pixel mask, and pixel gain to device 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_bad, bad, N_PIXELS * sizeof(short), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_gain, gain, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[N_STREAMS];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < N_STREAMS; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int evt = 0; evt < nEvents; evt++) {
    // Each event is divided into 32 streams
    int evtOffset = evt * N_PIXELS;
    for (int s=0; s < N_STREAMS; s++){
      // For copying data in, the offset is calculated from evt#
      int streamSize = ceil( (double) N_PIXELS / N_STREAMS );
      int offset = evtOffset + (s * streamSize);
      int streamBytes = streamSize * sizeof(float);   
      int gridSize = ceil(  (double) streamSize / blockSize );             

      checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                                 streamBytes, hipMemcpyHostToDevice,
                                 stream[s]) );

      // calibration kernels
      kernel<<<gridSize, blockSize, 0, stream[s]>>>(d_a, offset, d_dark, d_bad, cmmThr, streamSize, d_blockSum, d_cnBlockSum);
    
      // Common mode kernel reduce blockSum to sectorSum
      // We use 388 threads to reduce 388 blockSum (or sum of each row)
      // to a sector sum. No. of blocks is then equal to the no. of events.
      int cmmOffset = (evt * N_ROWS * N_STREAMS) + (s * N_ROWS);
      common_mode<<<1, N_ROWS, 0, stream[s]>>>(d_blockSum, d_cnBlockSum, d_sectorSum, d_cnSectorSum, cmmOffset);
      common_mode_apply<<<gridSize, blockSize, 0, stream[s]>>>(d_a, d_sectorSum, d_cnSectorSum, d_gain, offset); 

      // peakFinder kernels
      int filterOffset = (evt * FILTER_PATCH_PER_SECTOR * N_STREAMS) + (s * FILTER_PATCH_PER_SECTOR);
      filterByThrHigh_v2<<<FILTER_PATCH_PER_SECTOR, FILTER_THREADS_PER_PATCH, 0, stream[s]>>>(d_a, d_centers, filterOffset);
      
      // floodFill kernel is activiated by sending 64 threads to work
      // on each center.
      int ffOffset = (evt * nCentersPerSector * N_STREAMS) + (s * nCentersPerSector);
      floodFill_v2<<<nCentersPerSector, FF_LOAD_THREADS_PER_CENTER, 0, stream[s]>>>(d_a, d_centers, d_peaks, d_conmap, ffOffset);

      // copy data out
      checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[s]) );

      // copy peaks out
      checkCuda( hipMemcpyAsync(&peaks[ffOffset], &d_peaks[ffOffset],
                               nCentersPerSector * sizeof(Peak), hipMemcpyDeviceToHost,
                               stream[s]) ); 
    }
  }
  hipProfilerStop(); 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("GPU Calculation\n");
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("GPU Calibrated   : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("CPU Calibrated   : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", calib[0], calib[1], calib[2],calib[N_PIXELS-3], calib[N_PIXELS-2], calib[N_PIXELS-1]);
  printf("Differences      : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0]-calib[0], a[1]-calib[1], a[2]-calib[2], a[n-3]-calib[N_PIXELS-3], a[n-2]-calib[N_PIXELS-2], a[n-1]-calib[N_PIXELS-1]);
  printf("  max error      : %e\n", maxError(a, calib, nEvents));
     
  //write_file("calc_calib.txt", a, n);

  /*hipMemcpy(sectorSum, d_sectorSum, nSectors * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < nSectors; i++) {
    printf("i=%d sectorSum[i]=%f\n", i, sectorSum[i]);
  }*/
  
  /*int cnNonZeroCenters = 0;
  checkCuda( hipMemcpy(centers, d_centers, nCenters * sizeof(uint), hipMemcpyDeviceToHost) );
  for (int i=0; i < nCentersPerEvent * nEvents; i++){
    if (centers[i] != 0){
      int sectorId1 = (float) centers[i] / SECTOR_SIZE;
      int sectorId2 = (float) i / (nCentersPerSector);
      printf("i: %d, centers[i]:%d sectorByPixel: %d val: %6.2f sectorByIndex:%d\n", i, centers[i], a[centers[i]], sectorId1, sectorId2);
      cnNonZeroCenters++;
    }
  }
  printf("Total non zero centers: %d\n", cnNonZeroCenters);*/

  printf("nCenters: %d, nPeaks: %d, FFP_SECTOR: %d, W: %d, H: %d, nSectors: %d\n", nCenters,
            nPeaks, FILTER_PATCH_PER_SECTOR, FILTER_PATCH_WIDTH, FILTER_PATCH_HEIGHT, nSectors);

  int cnValidPeaks = 0;
  for (int i=0; i < nPeaks; i++) {
    if (peaks[i].valid) {
      //printf("i: %d, evt: %d, sector: %d, row: %d, col: %d\n", i, (int)peaks[i].evt, (int)peaks[i].seg, (int)peaks[i].row, (int)peaks[i].col);
      cnValidPeaks++;
    }
  }
  printf("nValidPeaks: %d\n", cnValidPeaks);
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < N_STREAMS; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  hipHostFree(raw);
  hipFree(d_dark);
  hipHostFree(dark);
  hipFree(d_gain);
  hipHostFree(gain);
  hipFree(d_calib);
  hipHostFree(calib);
  hipFree(d_blockSum);
  hipHostFree(blockSum);
  hipFree(d_cnBlockSum);
  hipHostFree(cnBlockSum);
  hipFree(d_sectorSum);
  hipHostFree(sectorSum);
  hipFree(d_cnSectorSum);
  hipFree(d_centers);
  hipHostFree(centers);
  hipFree(d_peaks);
  hipHostFree(peaks);
  hipFree(d_conmap);
  
  hipDeviceReset();

  return 0;
}
